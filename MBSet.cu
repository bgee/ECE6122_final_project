#include "hip/hip_runtime.h"
/* 
 * File:   MBSet.cu
 * 
 * Created on June 24, 2012
 * 
 * Purpose:  This program displays Mandelbrot set using the GPU via CUDA and
 * OpenGL immediate mode.
 * 
 */

#include <iostream>
#include <stack>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "Complex.cu"
#include <fstream>

#include <GL/freeglut.h>

// Size of window in pixels, both width and height
#define WINDOW_DIM            512
#define DIFF                  3.0
#define THREADS_PER_BLOCK     32

using namespace std;

// Initial screen coordinates, both host and device.
Complex minC(-2.0, -1.2);
Complex maxC(1.0, 1.8);
Complex* dev_minC;
Complex* dev_maxC;
const int maxIt = 2000; // Msximum Iterations

// Define the RGB Class
class RGB
{
public:
  RGB()
    : r(0), g(0), b(0) {}
  RGB(double r0, double g0, double b0)
    : r(r0), g(g0), b(b0) {}
public:
  double r;
  double g;
  double b;
};

RGB* colors = 0; // Array of color values
RGB *h_results, *d_results, *h_colors, *d_colors;

int s_x, s_y, e_x, e_y;
bool select_ready = false;
bool clicked = false;
Complex *minArray, *maxArray;
int arrayCount = 0;

void InitializeColors()
{
  colors = new RGB[maxIt + 1];
  for (int i = 0; i < maxIt; ++i)
    {
      if (i < 5)
        { // Try this.. just white for small it counts
          colors[i] = RGB(1, 1, 1);
        }
      else
        {
          colors[i] = RGB(drand48(), drand48(), drand48());
        }
    }
  colors[maxIt] = RGB(); // black
}

void lineLoop(int x1, int y1, int x2, int y2)
{
  glBegin(GL_LINE_LOOP);
  glVertex2i(x1, y1);
  glVertex2i(x2, y1);
  glVertex2i(x2, y2);
  glVertex2i(x1, y2);
  glEnd();
}

void display(void)
{
  glClearColor(1.0f, 1.0f, 1.0f, 1.0f);
  glClear(GL_COLOR_BUFFER_BIT);
  glLoadIdentity();
  //gluLookAt(0.0, 0.0, 0.00000000000005, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0);
  // glTranslatef(WINDOW_DIM/2, WINDOW_DIM/2, 0);
  glOrtho(-WINDOW_DIM/2, WINDOW_DIM/2, -WINDOW_DIM/2, WINDOW_DIM/2, 0.0, 1.0);
  //glScalef(0.003, 0.003, 0);
  glBegin(GL_POINTS);
  for (int i = 0; i < WINDOW_DIM; i++){
    for (int j = 0; j < WINDOW_DIM; j++){
      RGB current = h_results[i*WINDOW_DIM+j];
      glColor3d(current.r, current.g, current.b);
      //glColor3d(0.5, 0.0, 0.0);
      //glVertex2f(500, 500);
      //cout << current.r << " " << current.g << " ";
      //cout << current.b << endl;
      glVertex2i(i - WINDOW_DIM/2, j - WINDOW_DIM/2);
      //glVertex2i(-i, -j);
      //glVertex2i(i, -j);
      //glVertex2i(-i, j);
      //cout << i-WINDOW_DIM/2 << " " << j - WINDOW_DIM/2 << endl;
    }
  }
  glEnd();
  if (select_ready || clicked){
    glColor3d(1.0, 0.0, 0.0);
    lineLoop(s_x - WINDOW_DIM/2, -s_y+WINDOW_DIM/2, e_x-WINDOW_DIM/2,
	    -e_y+WINDOW_DIM/2);
  }
  glFinish();
  glutSwapBuffers();
}

void reshape(int w, int h)
{
  glMatrixMode(GL_MODELVIEW);
  glLoadIdentity();
}

/*
void setupSet(Complex* set)
{
  for (int i = 0; i < WINDOW_DIM; i++){
    for (int j = 0; j < WINDOW_DIM; j++){
      *(set+i*WINDOW_DIM + j) = minC + Complex(float(i)/float(WINDOW_DIM)*DIFF,
					       float(j)/float(WINDOW_DIM)*DIFF);
    }
  }
}
*/

__global__ void computeSingle(RGB *d_results, RGB *d_colors, double r,
			      double i, double diff)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  //printf("%d\n", index);
  int row = index / WINDOW_DIM;
  int col = index % WINDOW_DIM;
  Complex current = Complex(r + diff*double(row)/double(WINDOW_DIM),
			    i + diff*double(col)/double(WINDOW_DIM));
  Complex c = Complex(current);
  int count = -1;
  while ((count <= (maxIt+1)) && (current.magnitude2() < 4)){
    current = current * current + c;
    count++;
  }
  //if (count == 0){printf("count zero %d %d\n", row, col);}
  if (count == -1){
    d_results[index].r = 1.0;
    d_results[index].g = 1.0;
    d_results[index].b = 1.0;
  }
  else {
    d_results[index].r = d_colors[count].r;
    d_results[index].g = d_colors[count].g;
    d_results[index].b = d_colors[count].b;
  }
}


/*__global__ void computeSet(RGB *d_results, RGB *d_colors)
{
  int b = WINDOW_DIM;
  for (int i = 0; i < b; i++){
    for (int j = 0; j < b; j++){
      //printf("abc %d, %d\n", i, j);
      int count = -1;
      Complex current = Complex(-2.0 + DIFF*double(i)/double(WINDOW_DIM),
			        -1.2 + DIFF*double(j)/double(WINDOW_DIM));
      Complex c = Complex(current);
      //printf("before while\n");
      while ((count < 2000) && (current.magnitude2() < 4)){
	current = current * current + c;
	count++;
      }
      //printf("after while%d\n", count);
      d_results[i*WINDOW_DIM + j].r = d_colors[count].r;
      d_results[i*WINDOW_DIM + j].g = d_colors[count].g;
      d_results[i*WINDOW_DIM + j].b = d_colors[count].b;
      //printf("%d, %d, %d\n", i, j, count);
    }
  }
  for (int i = 0; i < b; i++){
    for (int j = 0; j < b; j++){
      if (d_results[i*b + j].r != 0.0){
	//printf("%d, %d ", i, j);
      }
    }
  }
  printf("\n");
  }*/

void mouse(int button, int state, int x, int y)
{
  if (button == GLUT_LEFT_BUTTON && state == GLUT_DOWN){
    s_x = x;
    s_y = y;
    clicked = true;
    cout << "clicked true" << button << GLUT_LEFT_BUTTON << " "
	 << state << GLUT_DOWN << endl;
  }

  else if (button == GLUT_LEFT_BUTTON && state == GLUT_UP){
    e_x = x;
    e_y = e_x - s_x + s_y;
    select_ready = true;
    clicked = false;
    glutPostRedisplay();
    double delta = maxC.r - minC.r;
    if (e_x < s_x){
      int temp_x = s_x;
      int temp_y = s_y;
      s_x = e_x;
      s_y = e_y;
      e_x = temp_x;
      e_y = temp_y;
    }
    double start = delta*((double) (s_x)) / ((double) WINDOW_DIM);
    int min_x = (s_x < e_x) ? s_x : e_x;
    int min_y = (s_y > e_y) ? s_y : e_y;
    double start_x = delta*((double) (min_x)) / ((double) WINDOW_DIM);
    double start_y = delta - delta*((double) (min_y)) / ((double) WINDOW_DIM);
    int max_x = (s_x > e_x) ? s_x : e_x;
    int max_y = (s_y < e_y) ? s_y : e_y;
    double end_x = delta - delta*((double) (max_x)) / ((double) WINDOW_DIM);
    double end_y = delta*((double) (max_y)) / ((double) WINDOW_DIM);
    minArray[arrayCount] = minC;
    maxArray[arrayCount] = maxC;
    arrayCount++;
    minC = Complex(minC.r + start_x,
		   minC.i + start_y);
    double end = delta*(1.0 - ((double) (e_x)) / ((double) WINDOW_DIM));
    maxC = Complex(maxC.r - end_x, end_y);
    cout << "------------" << min_x << " " << min_y << " " << max_x << " "
	 << max_y << endl;
    cout << "------------" << start_x << " " << start_y << " " << end_x
	 << " " << end_y << endl;
    cout << "before computeSingle" << endl;
    computeSingle<<<WINDOW_DIM*WINDOW_DIM/THREADS_PER_BLOCK,
      THREADS_PER_BLOCK>>>(d_results, d_colors, minC.r, minC.i, maxC.r-minC.r);
    hipMemcpy(h_results, d_results, WINDOW_DIM*WINDOW_DIM*sizeof(RGB),
	       hipMemcpyDeviceToHost);
    cout << s_x << " " << s_y << " " << e_x << " " << e_y << endl;
    cout << minC.r << " " << minC.i << " aaaa" << endl;
    cout << maxC.r << " " << maxC.i << " bbbb " << end << endl;
    select_ready = false;
    glutPostRedisplay();
    //display();
  }
  /*if (clicked && button == GLUT_LEFT_BUTTON && state == GLUT_DOWN){
    cout << "clicked" << endl;
    e_x = x;
    e_y = e_x - s_x + s_y;
    glutPostRedisplay();
    }*/
  cout << s_x << " " << s_y << " " << e_x << " " << e_y << endl;
}

void motion(int x, int y)
{
  if (clicked){
    e_x = x;
    e_y = e_x - s_x + s_y;
    glutPostRedisplay();
  }
}

void keyboard(unsigned char key, int x, int y)
{
  if (key == 'b' && arrayCount > 0){
    minC = minArray[arrayCount-1];
    maxC = maxArray[arrayCount-1];
    arrayCount--;
    computeSingle<<<WINDOW_DIM*WINDOW_DIM/THREADS_PER_BLOCK,
      THREADS_PER_BLOCK>>>(d_results, d_colors, minC.r, minC.i, maxC.r-minC.r);
    hipMemcpy(h_results, d_results, WINDOW_DIM*WINDOW_DIM*sizeof(RGB),
	       hipMemcpyDeviceToHost);
    glutPostRedisplay();
  }
}

int main(int argc, char** argv)
{
  // debug file to output RGB
  ofstream r, g, b;
  r.open("r.csv");
  g.open("g.csv");
  b.open("b.csv");
  // Initialize OPENGL here
  // Set up necessary host and device buffers
  // set up the opengl callbacks for display, mouse and keyboard
  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH | GLUT_MULTISAMPLE);
  glutInitWindowSize(WINDOW_DIM, WINDOW_DIM);
  glutInitWindowPosition(100, 100);
  glutCreateWindow("MB Set");
  glClearColor(0.0, 0.0, 0.0, 1.0);
  cout << "here" << endl;
  h_colors = colors;
  // allocate space for pixels
  hipMalloc((void **)&d_results, WINDOW_DIM*WINDOW_DIM*sizeof(RGB));
  hipMalloc((void **)&d_colors, (maxIt+1)*sizeof(RGB));
  h_results = new RGB[WINDOW_DIM*WINDOW_DIM*sizeof(RGB)];
  // initialize array for min and max
  minArray = (Complex*)malloc(100*sizeof(Complex));
  maxArray = (Complex*)malloc(100*sizeof(Complex));

  arrayCount = 0;
  cout << "after hipMalloc" << endl;
  InitializeColors();
  ofstream colo;
  colo.open("colo.csv");
  for (int i = 0; i < (maxIt+1); i++){
    colo << colors[i].r << " " << colors[i].g << " " << colors[i].b << endl;
  }
  colo.close();
  cout << "after colors" << endl;
  hipMemcpy(d_colors, colors, (maxIt + 1)*sizeof(RGB),
	     hipMemcpyHostToDevice);
  cout << "after hipMemcpy" << endl;
  // allocate space for complex matrix
  // set = new Complex[WINDOW_DIM * WINDOW_DIM];
  // Calculate the interation counts
  cout << "before comuteSet" << endl;
  //computeSet<<<1, 1>>>(d_results, d_colors);
  computeSingle<<<WINDOW_DIM*WINDOW_DIM/THREADS_PER_BLOCK,
    THREADS_PER_BLOCK>>>(d_results, d_colors, minC.r, minC.i, maxC.r-minC.r);
  cout << "after computeSet" << endl;
  hipMemcpy(h_results, d_results, WINDOW_DIM*WINDOW_DIM*sizeof(RGB),
	     hipMemcpyDeviceToHost);
  cout << "after copy result" << endl;
  for (int i = 0; i < WINDOW_DIM*WINDOW_DIM; i++){
    //h_results[i] = RGB(1, 1, 1);
    //cout << h_results[i].r << " ";
  }
  for (int i = 0; i < WINDOW_DIM; i++){
    for (int j = 0; j < WINDOW_DIM; j++){
      if (h_results[i*WINDOW_DIM + j].r != 0.0){
	//cout << i << " " << j << " ";
      }
      RGB c = h_results[i*WINDOW_DIM + j];
      r << c.r << ",";
      g << c.g << ",";
      b << c.b << ",";
    }
    r << "\n"; g << "\n"; b << "\n";
  }
  r.close(); g.close(); b.close();
  cout << endl;
  //hipFree(d_results);
  glutDisplayFunc(display);
  glutMouseFunc(mouse);
  glutMotionFunc(motion);
  glutKeyboardFunc(keyboard);
  // Grad students, pick the colors for the 0 .. 1999 iteration count pixels

  glutMainLoop(); // THis will callback the display, keyboard and mouse
  return 0;

}
